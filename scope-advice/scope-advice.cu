#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.

 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

#include <algorithm>
#include <assert.h>
#include <fstream>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <tuple>
#include <unistd.h>

//#define DEBUG_OUT

#include "helper.h"

void handle_memory_access(mem_access_t *ma, int tid) {
    uint64_t addr = ma->addr, md_offset;
    md_offset = (addr / GRAN) % host_metadata_len;
    bool done = false;
    m_packets.fetch_add(1);
    unsigned delay = HOST_BASE_DELAY;
    while (!done) {
        uint64_t expected(access_map[md_offset].load());
        uint64_t desired(LOCKED);
        if (expected == desired) {
            /* someone holds the lock --- backoff */
            backoff(delay);
            continue;
        }

        if (access_map[md_offset].compare_exchange_strong(expected, desired)) {
            std::vector<uint64_t> *s;
            /* Zero initialized, if not, meaning some address present! */
            if (expected == 0) {
                s = new std::vector<uint64_t>;
                // (*s).reserve(UNIQ_THRESHOLD);
            } else {
                s = (std::vector<uint64_t>*)expected;
            }

            (*s).push_back(ma->info);
            expected = (uint64_t)s;
            if ((*s).size() > UNIQ_THRESHOLD) {
                pthread_mutex_lock(&async_lock);
                /* Insert offset */
                cleaner_queue.insert(md_offset);
                pthread_mutex_unlock(&async_lock);
            }
            /* Atomically write to it! */
            access_map[md_offset].exchange(expected);
            done = true;
        } else {
            /* someone got the lock --- backoff */
            backoff(delay);
        }
    }
}

/* All conditions have to be met for this to work:
   1. If multi_block
   2. If there are stores
   3. If there are weak operations, or cta scoped operations
If yes to all questions, all relevant epochs in access_map have to be utilized.
    for store epochs, next one is useful, aka, release operation
    for load epochs, previous one is useful, aka, acquire operation. */
void process_access_info(int tid, allocation record) {
    uint64_t per_thread, sidx, eidx;
    /* Divide addresses in record into NUM_THREADS portions */
    per_thread = (record.bound - record.base) / NUM_THREADS;

    /* start and end address */
    sidx = tid * per_thread + record.base;
    if (tid == NUM_THREADS - 1)
        eidx = record.bound;
    else
        eidx = (tid + 1) * per_thread + record.base;

    /* traverse across allocated addresses */
    for (uint64_t addr = sidx; addr < eidx; addr += GRAN) {
        uint64_t i = (addr / GRAN) % host_metadata_len;

        uint64_t md_offset = (addr / GRAN) % device_arguments.length;
        uint64_t md = device_arguments.memory_meta[md_offset];
        // print_md(md, addr);
        if (getBit(md, POS_MB) && getBit(md, POS_ST)) {
            if (DO_STREAM) {
                /* get content from stream_meta */
                uint64_t count = getBits(md, POS_CNT, SZ_CNT);
                for (uint64_t j = 0; j < count && j < NUM_STREAM_TRACES; j++) {
                    uint64_t _offset = md_offset + j * device_arguments.length;
                    uint64_t trace = device_arguments.stream_meta[_offset];
                    process_trace(trace);
                }
            }
            /* Traverse the set! */
            uint64_t possible_vector = access_map[i].load();
            if (possible_vector != 0) {
                std::vector<uint64_t> *s = (std::vector<uint64_t>*)possible_vector;
                for (auto trace : *s) {
                    process_trace(trace);
                }
            }
        }
    }
}

/* iterate over all allocations */
void iterate_allocations(int tid) {
    if (DO_ANALYZE) {
        for (auto each: allocation_records) {
            process_access_info(tid, each);
        }
    }
}

void worker(int id) {

    int jobs_handled = 0;
    while (1) {

        int i = JOB_NONE;
        pthread_mutex_lock(&job_lock);
        if (job_queue.size() != 0) {
            i = job_queue.back();
            job_queue.pop_back();
        }
        pthread_mutex_unlock(&job_lock);

        if (i != JOB_NONE) {
            channel_t *chan = (channel_t*)jobs[i].buffer;
            /* Each worker-thread figures out their own content */
            uint32_t num_entries = jobs[i].job_amount / sizeof(channel_t);
            uint32_t start_entry = 0;
            // printf("%d: Got job of size: %u (%uB)\n", id, num_entries, jobs[i].job_amount);

            while (start_entry < num_entries) {
                channel_t *c = &chan[start_entry];
                // printf("%d: processing %u @%p\n", id, start_entry, c);
                if (c->type == TYPE_MEM) {
                    handle_memory_access(&c->ma, id);
                }
                start_entry += 1;
            }

            /* Push back to free queue */
            pthread_mutex_lock(&free_lock);
            free_queue.push_back(i);
            pthread_mutex_unlock(&free_lock);

            // printf("%d: %d done, waiting .... status\n", id, i);
            jobs_handled += 1;
        } else if (last_job.load() == JOB_NONE) {
            /* no job in queue and last job seen by distributor, break */
            // printf("%d: all jobs done ... exiting\n", id);
            break;
        }
    }

    // printf("%d: finished %d jobs ... moving to detection. Wait till dedup reaches barrier\n", id, jobs_handled);
    pthread_barrier_wait(&barrier);
    // avoid races on 'detection' var ... make only 1 thread update it
    if (id == 0)
        detection.start();
    /* Parallelize detection logic */
    iterate_allocations(id);
    // jobs being equally allocated among workers, they are expected to finish together
    if (id == 0)
        detection.end();
}

void set_meta(int id, allocation record) {
    /* only 1 thread does this! Avoid launching too many tasks! */
    if (id != 0)
        return;

    uint64_t sidx, eidx, size, length = device_arguments.length;
    uint32_t *base = device_arguments.memory_meta, *end;
    sidx = (record.base / GRAN) % length;
    eidx = (record.bound / GRAN) % length;
    if (sidx > eidx) {
        /* first half */
        size = (uint64_t)(base + eidx) - (uint64_t)base;
        hipMemsetAsync(device_arguments.memory_meta, 0, size, stream);
        end = base + length;
        /* second half */
        size = (uint64_t)end - (uint64_t)(base + sidx);
        hipMemsetAsync(device_arguments.memory_meta + sidx, 0, size, stream);
    } else {
        size = record.bound - record.base;
        hipMemsetAsync(device_arguments.memory_meta + sidx, 0, size, stream);
    }
}

void *async_zero(void *arg) {
    thread_data_t *data = (thread_data_t *)arg;
    int tid = data->tid;
    uint64_t per_thread, s_addr, e_addr, sidx, eidx, size;

    /* Wait till the instrumenation completes. Syncing with main thread (which does instrumenttion) */
    pthread_barrier_wait(&barrier);

    for (auto record: allocation_records) {
        per_thread = (record.bound - record.base) / NUM_THREADS;

        /* start and end address */
        s_addr = tid * per_thread + record.base;
        if (tid == NUM_THREADS - 1)
            e_addr = record.bound;
        else
            e_addr = (tid + 1) * per_thread + record.base;

        sidx = (s_addr / GRAN) % host_metadata_len;
        eidx = (e_addr / GRAN) % host_metadata_len;
        if (sidx > eidx) {
            std::atomic<uint64_t> *end = access_map + host_metadata_len;
            /* first half */
            size = (uint64_t)(access_map + eidx) - (uint64_t)access_map;
            memset((uint64_t*) access_map, 0, size);
            /* second half */
            size = (uint64_t)end - (uint64_t)(access_map + sidx);
            memset((uint64_t*) access_map + sidx, 0, size);
        } else {
            memset((uint64_t*) access_map + sidx, 0, e_addr - s_addr);
        }
        set_meta(tid, record);
    }
    /* Wait till setup is complete. Syncing with min thread (which does device metadata setup) */
    pthread_barrier_wait(&barrier);

    worker(tid);
    pthread_exit(NULL);
}

void *deduplicate(void *arg) {

    unsigned long long cleaner_jobs = 0, cleaned = 0;
    while (1) {
        if (last_job.load() == JOB_NONE)
            break;

        if (cleaner_queue.size() == 0)
            continue;

        pthread_mutex_lock(&async_lock);
        cleaner_jobs++;
        std::unordered_set<uint64_t> l_job = cleaner_queue;
        cleaner_queue.clear();
        pthread_mutex_unlock(&async_lock);
        // printf("cleaner: got job of size: %lu\n", l_job.size());
        for (uint64_t s: l_job) {
            /* Set has offsets into access_map, no need to recalculate */
            bool done = false;
            unsigned delay = HOST_BASE_DELAY;
            while (!done) {
                uint64_t expected(access_map[s].load());
                uint64_t desired(LOCKED);
                if (expected == desired) {
                    /* someone holds the lock --- backoff */
                    backoff(delay);
                    continue;
                }

                if (access_map[s].compare_exchange_strong(expected, desired)) {
                    std::vector<uint64_t> *se = (std::vector<uint64_t>*)expected;

                    std::sort((*se).begin(), (*se).end());
                    (*se).erase(std::unique((*se).begin(), (*se).end()), (*se).end());

                    expected = (uint64_t)se;
                    cleaned++;
                    /* Atomically write to it! */
                    access_map[s].exchange(expected);
                    done = true;
                } else {
                    /* someone got the lock --- backoff */
                    backoff(delay);
                }
            }
        }
    }
    /* Participate in the barrier. Syncing with worker threads (waiting after processing all packets) */
    pthread_barrier_wait(&barrier);
    // printf("[Cleaner] Clean jobs: %llu, Cleaned offsets: %llu .... exiting\n", cleaner_jobs, cleaned);
    pthread_exit(NULL);
}

void prefetch_device_metadata() {
    /* use stream to prefetch fence content */
    hipMemPrefetchAsync(device_arguments.fence_meta, sizeof(uint32_t) * kernel_dimension.warpsInGrid * epoch, hipCpuDeviceId, stream);
    uint64_t base, bound, sidx, eidx, size;
    /* prefetch memory metadata */
    for (auto each: allocation_records) {
        base = each.base;
        bound = each.bound;

        sidx = (base / GRAN) % device_arguments.length;
        eidx = (bound / GRAN) % device_arguments.length;
        /* if eidx < sidx, requires roundabout */
        if (eidx < sidx) {
            uint32_t* end = device_arguments.memory_meta + host_metadata_len;
            size = (uint64_t)end - (uint64_t)(device_arguments.memory_meta + sidx);
            hipMemPrefetchAsync(device_arguments.memory_meta + sidx, size, hipCpuDeviceId, stream);
            size = (uint64_t)(device_arguments.memory_meta + eidx) - (uint64_t)device_arguments.memory_meta;
            hipMemPrefetchAsync(device_arguments.memory_meta, size, hipCpuDeviceId, stream);
        } else {
            hipMemPrefetchAsync(device_arguments.memory_meta + sidx, bound - base, hipCpuDeviceId, stream);
        }
    }
}

void *distributor(void *) {
    while(recv_thread_started) {

        /* Spin-lock to check size */
        pthread_mutex_lock(&free_lock);
        /* It's ok to do this while holding lock as the procedure is short */
        if (free_queue.size() != 0) {
            int i = free_queue.back();
            uint32_t num_recv_bytes = 0;
            /* Boss thread --- waits for generated data to process */
            if (recv_thread_receiving && (num_recv_bytes = channel_host.recv(jobs[i].buffer, CHANNEL_SIZE)) > 0) {
                message_passes += 1;
                /* Don't have to reset this! */
                if (message_passes == 1)
                    message.start();

                /* Write job information */
                jobs[i].job_amount = num_recv_bytes;
                // printf("Boss: set up job %d\n", i);

                pthread_mutex_lock(&job_lock);
                /* Remove from free queue */
                free_queue.pop_back();
                /* Push to job queue */
                job_queue.push_back(i);
                pthread_mutex_unlock(&job_lock);

                /* Check if it was last message */
                char *recv = jobs[i].buffer;
                recv = recv + num_recv_bytes - sizeof(channel_t);
                channel_t *possible_last_message = (channel_t*)recv;
                if (possible_last_message->type == TYPE_INV) {
                    /* distributor can now return, nothing more to do */
                    recv_thread_receiving = false;
                    last_job.exchange(JOB_NONE);
                    /* No break as lock needs to be released */
                    message.end();
                }
            } else if (!recv_thread_receiving) {
                /* Re executing instrumented kernel can generate messages. If not processed
                   can block the kernel. Process them by putting content in a dummy buffer. */
                num_recv_bytes = channel_host.recv(&dummy_buffer, CHANNEL_SIZE);
            }
        }
        /* Release lock */
        pthread_mutex_unlock(&free_lock);
    }
    // prefetch_device_metadata();
    pthread_exit(NULL);
}


__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is
     * completed */
    channel_t c;
    c.type = TYPE_INV;
    device_arguments.channel_dev->push(&c, sizeof(channel_t));

    /* flush channel */
    device_arguments.channel_dev->flush();
}


/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        uint64_t base_addr = nvbit_get_func_addr(f);
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
        if (verbose) {
            printf("Inspecting function %s at address 0x%lx\n", nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
        }

        uint32_t cnt = 0;
        /* iterate on all the static instructions in the function */
        bool memory_between = false;
        /* Inserting one for KERNEL_BEGIN */
        fence_map[-1] = new fence_info(-1, !memory_between);
        for (auto instr : instrs) {
            uint64_t offset = instr->getOffset();
            if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
                (instr->getMemorySpace() == InstrType::MemorySpace::NONE && !isBarrier(instr) &&
                !isFence(instr) && !(isWarpBar(instr) && check_its))) {
                cnt++;
                continue;
            }
            
            cnt++;
            if (verbose) {
                instr->printDecoded();
            }

           if (isBarrier(instr)) {
                if (!memory_between) {
                    /* Make previous fence a candidate for redundancy! Case where fence comes before barrier */
                    if (epoch != 0)
                        fence_map[epoch - 1]->is_redundant = true;
                }
                memory_between = false;
                continue;
            }

            /* Need only device scope for now, not useful to keep track of block scope */
            if(isFence(instr) && getScope(instr) == SCOPE_GPU) {
                /* Add some instrumentation information! */
                nvbit_insert_call(instr, "instrument_fence", IPOINT_BEFORE);
                /* predicate value */
                nvbit_add_call_arg_guard_pred_val(instr);
                /* epoch value */
                volatile int l_epoch = epoch;
                nvbit_add_call_arg_const_val32(instr, (uint32_t)l_epoch);
                /* pointer to location with device_arguments struct */
                nvbit_add_call_arg_const_val64(instr, (uint64_t)&device_arguments);

                /* Maintain info for making suggestions later */
                uint64_t addr =  base_addr + offset;
                if (fence_to_lineinfo_map.find(addr) == fence_to_lineinfo_map.end()) {
                    id_to_fence_map[epoch] = addr;

                    char* file_name;
                    char* dir_name;
                    uint32_t line;
                    bool avail = nvbit_get_line_info(ctx, f, instr->getOffset(), &file_name, &dir_name, &line);
                    std::string output;
                    if(avail)
                        output = std::string(file_name) + " - Kernel " + std::string(nvbit_get_func_name(ctx, f)) + ": Line " + std::to_string(line) + "    " + instr->getSass();
                    else
                        output = std::string(instr->getSass()) + " - Kernel " + std::string(nvbit_get_func_name(ctx, f)) + ": Sass offset " + std::to_string(instr->getOffset());
                    fence_to_lineinfo_map[addr] = output;

                    /* Information for type of OS if memory_between -> not_redundant */
                    fence_map[epoch] = new fence_info(epoch, !memory_between);
                    memory_between = false;
                    /* This epoch will be used during memory instrumentation */
                    epoch += 1;
                }
                continue;
            }

            int mref_idx = 0;
            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const InstrType::operand_t *op = instr->getOperand(i);

                if (op->type == InstrType::OperandType::MREF && 
                    (instr->getMemorySpace() == InstrType::MemorySpace::GENERIC
                    || instr->getMemorySpace() == InstrType::MemorySpace::GLOBAL)) {
                    /* insert call to the instrumentation function with its
                     * arguments */
                    nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                    /* predicate value */
                    nvbit_add_call_arg_guard_pred_val(instr);
                    /* memory reference 64 bit address */
                    nvbit_add_call_arg_mref_addr64(instr, mref_idx);
                    /* information about memory operation */
                    nvbit_add_call_arg_const_val32(instr, getScope(instr) | getLoadStoreMask(instr));
                    /* A precaution to copy to local */
                    volatile int l_epoch = epoch;
                    nvbit_add_call_arg_const_val32(instr, l_epoch);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val32(instr, (uint32_t)instr->getSize());
                    /* add instruction value */
                    int l_counter = static_counter;
                    nvbit_add_call_arg_const_val32(instr, (uint32_t)l_counter);
                    static_counter += 1;
                    /* pointer to location with device_arguments struct */
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)&device_arguments);
                    mref_idx++;
                    memory_between = true;
                } else if (op->type == InstrType::OperandType::MREF &&
                    instr->getMemorySpace() == InstrType::MemorySpace::SHARED) {
                    memory_between = true;
                }
            }
        }
        /* Inserting final one, KERNEL_END */
        fence_map[epoch] = new fence_info(epoch, !memory_between);
    }
}


void set_allocations(nvbit_api_cuda_t cbid, void *params) {
    uint64_t local_base, local_bound;
    setup.start();
    switch(cbid) {
        case API_CUDA_cuMemAlloc_v2: {
            cuMemAlloc_v2_params *p1 = (cuMemAlloc_v2_params *)params;
            local_base = (uint64_t)*p1->dptr;
            local_bound = (uint64_t)*p1->dptr + p1->bytesize;
            break;
        }
        case API_CUDA_cuMemAllocManaged: {
            cuMemAllocManaged_params *p2 = (cuMemAllocManaged_params *)params;
            local_base = (uint64_t)*p2->dptr;
            local_bound = (uint64_t)*p2->dptr + p2->bytesize;
            break;
        }
        case API_CUDA_cuMemAllocPitch_v2: {
            cuMemAllocPitch_v2_params *p3 = (cuMemAllocPitch_v2_params *)params;
            local_base = (uint64_t)*p3->dptr;
            local_bound = (uint64_t)*p3->dptr + (p3->WidthInBytes * p3->Height);
            break;
        }
        case API_CUDA_cuModuleGetGlobal_v2: {
            cuModuleGetGlobal_v2_params_st *p4 = (cuModuleGetGlobal_v2_params_st *)params;
            local_base = (uint64_t)*p4->dptr;
            /* HACK: size for global allocations not available. Correct way is (uint64_t)*(p4->bytes) */
            local_bound = (uint64_t)*p4->dptr + NUM_THREADS * GRAN * 2;
            break;
        }
        default:
            return;
    }
    allocation_records.emplace_back(local_base, local_bound);
    app_mem += (local_bound - local_base);
    // for in-GPU metadata, 4B per each 4B addr
    meta_mem += (local_bound - local_base);
    // for in-GPU trace, NUM_STREAM_TRACES * 4B per each 4B addr (only when enabled)
    if (DO_STREAM)
        meta_mem += ((local_bound - local_base) * NUM_STREAM_TRACES);
    setup.end();
}


void set_dimension(cuLaunchKernel_params *p) {
    kernel_dimension.blockDim = p->blockDimX * p->blockDimY * p->blockDimZ;
    kernel_dimension.warpsPerBlock = roundUp(kernel_dimension.blockDim, WARP_SIZE);
    kernel_dimension.gridDim = p->gridDimX * p->gridDimY * p->gridDimZ * kernel_dimension.blockDim;
    kernel_dimension.warpsInGrid = roundUp(kernel_dimension.gridDim, WARP_SIZE);
    /* Set information that can be sent to instrumented device functions */
    device_arguments.threads_per_block = kernel_dimension.blockDim;
    device_arguments.threads = kernel_dimension.gridDim;
}



void set_sampling_meta() {
    if (!DO_SAMPLING)
        return;

    /* Requires kernel_dimension to be set! */
    srand(time(0));
    uint64_t blocks = (kernel_dimension.gridDim / kernel_dimension.blockDim) + 1;
    /* Set up metadata for instr-thread level sampling, no need to instrument this, skipping */
    skip_flag = true;
    hipMallocManaged((void**)&device_arguments.random_meta, sizeof(char) * blocks);
    samp_mem += sizeof(char) * blocks;
    for (uint64_t i = 0; i < blocks; i++) {
        /* create a random number between SAMP_BASE and PER_THREAD_PER_INSTR */
        device_arguments.random_meta[i] = rand() % (PER_THREAD_PER_INSTR - SAMP_BASE + 1) + SAMP_BASE;
    }
    hipMallocManaged((void**)&device_arguments.sampling_meta, sizeof(char) * kernel_dimension.gridDim * static_counter);
    samp_mem += sizeof(char) * kernel_dimension.gridDim * static_counter;
    // memset async as the driver launches the kernel after these operations are over
    hipMemsetAsync(device_arguments.sampling_meta, 0, sizeof(char) * kernel_dimension.gridDim * static_counter, stream);
    skip_flag = false;
}


void set_fence_meta() {
    int entries = kernel_dimension.warpsInGrid;
    skip_flag = true;
    // Should consider a different multiple here if the threads_per_block is not a multiple of WARP_SIZE
    hipMallocManaged((void**)&device_arguments.fence_meta, sizeof(uint32_t) * entries * epoch);
    fence_mem +=  sizeof(uint32_t) * entries * epoch;
    device_arguments.warps_per_grid = entries;
    skip_flag = false;
}

/*****************************************************
 *                                                   *
 *  NVBIT Instrumentation Interface Calls Below      *
 *                                                   *
******************************************************/


void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool (def = 0)");
    GET_VAR_INT(timeout, "TIMEOUT", 0, "Time in seconds after which to quit detection (0 = never; def = 0)");
    GET_VAR_INT(debug_out, "DEBUG", 0, "Output debug info (def = 0)");
    GET_VAR_STR(kernel_id, "KERNELID", "Specific kernel that needs to be traced (def = all)");
    GET_VAR_INT(instance, "INSTANCE", 1, "The dynamic instance of the KERNELID to be traced (def = first)");
    std::string pad(100, '-');
    printf ("%s\n", pad.c_str());
}


void nvbit_at_cuda_event (hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    /* returning from memory allocation APIs */
    if (is_exit && cbid == API_CUDA_cuMemAlloc_v2) {
        set_allocations(API_CUDA_cuMemAlloc_v2, params);
        return;
    } else if (is_exit && cbid == API_CUDA_cuMemAllocManaged) {
        set_allocations(API_CUDA_cuMemAllocManaged, params);
        return;
    } else if (is_exit && cbid == API_CUDA_cuMemAllocPitch_v2) {
        set_allocations(API_CUDA_cuMemAllocPitch_v2, params);
        return;
    } else if (is_exit && cbid == API_CUDA_cuModuleGetGlobal_v2) {
        set_allocations(API_CUDA_cuModuleGetGlobal_v2, params);
        return;
    }

    if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel ||
        cbid == API_CUDA_cuLaunchCooperativeKernel || cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz) {

        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
        if (!kernel_id.empty()) {
            // Check for no match. Skip the kernel
            if(strstr(nvbit_get_func_name(ctx, p->f), kernel_id.c_str()) == NULL)
                return;

            if (!is_exit) {
                // Given kernel, increase instance count. Only increment on entry
                kernel_instances += 1;
            }
            // If this is the suggested instance do it, else skip
            if (kernel_instances != instance) {
                // Disable instrumentation if the instance is not to be traced
                // Only do this after required instance is traced
                if (kernel_instances > instance) {
                    nvbit_enable_instrumented(ctx, p->f, false);
                }
                return;
            }
        }

        if (!is_exit) {
            instrumentation.start();
            instrument_function_if_needed(ctx, p->f);
            nvbit_enable_instrumented(ctx, p->f, true);
            instrumentation.end();
            setup.start();
            /* let dedup thread move ahead as well! */
            last_job.exchange(JOB_BEGIN);

            /* Start zeroing, this will wake up the workers, after instrumentation completes! */
            pthread_barrier_wait(&barrier);

            int nregs;
            CUDA_SAFECALL (hipFuncGetAttribute (&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

            int shmem_static_nbytes;
            CUDA_SAFECALL (hipFuncGetAttribute (&shmem_static_nbytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));
            printf ("Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
                "%d - shmem %d - cuda stream id %ld\n", nvbit_get_func_name(ctx, p->f),
                p->gridDimX, p->gridDimY, p->gridDimZ, p->blockDimX, p->blockDimY,
                p->blockDimZ, nregs, shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);

            /* Useful for calculation later */
            set_dimension(p);
            /* Information needed for implementing execution sampling */
            set_sampling_meta();
            /* initialize fence meta */
            set_fence_meta();

            /* Ensure that workers have completed zeroing! */
            pthread_barrier_wait(&barrier);

            /* sync zeroing */
            hipStreamSynchronize(stream);

            setup.end();
            kernel.start();
            /* Ensure that boss thread now starts listening for GPU jobs */
            recv_thread_receiving = true;
        } else {
            /* Removing this can cause trouble, as flush marker below must be set after kernel finishes */
            hipDeviceSynchronize ();
            hipError_t error = hipGetLastError ();
            if (error != hipSuccess) {
                printf ("CUDA error_%d: %s\n", error, hipGetErrorName (error));
                assert (false);
            }
            kernel.end();
            /* Will be launching a kernel from here, so skip all instrumentation of that one */
            skip_flag = true;

            flush_channel<<<1,1>>> ();
            hipDeviceSynchronize ();
            error = hipGetLastError ();
            if (error != hipSuccess) {
                printf ("CUDA error_%d: %s\n", error, hipGetErrorName (error));
                assert (false);
            }
            
            /* All good, restart instrumentation */
            skip_flag = false;
        }
    }
}


void nvbit_at_ctx_init (hipCtx_t ctx) {
    setup.start();
    if (!recv_thread_started) {
        /* Initialize job content */
        for (int i = 0; i < NUM_BUFFERS; i++) {
            jobs[i].job_amount = 0;
            jobs[i].buffer = (char *)malloc (CHANNEL_SIZE);
            free_queue.push_back(i);
        }

        /* Need not init this for every ctx, just once! */
        recv_thread_started = true;
        channel_host.init (0, CHANNEL_SIZE, &channel_dev, NULL);
        /* set up channel in device_arguments */
        device_arguments.channel_dev = &channel_dev;
        /* Init locks for job, free queues, async */
        pthread_mutex_init(&async_lock, NULL);
        pthread_mutex_init(&free_lock, NULL);
        pthread_mutex_init(&job_lock, NULL);
        /* Creates a barrier with workers + async_task amount of threads */
        pthread_barrier_init(&barrier, NULL, NUM_THREADS + 1);
        /* Create boss thread */
        int result = pthread_create (&recv_thread, NULL, distributor, NULL);
        /* Create cleaner thread */
        result = pthread_create (&async_task, NULL, deduplicate, NULL);
        for (int i = 0; i < NUM_THREADS; ++i) {
            thr_data[i].tid = i;
            /* Create multiple worker threads! */
            if ((result = pthread_create(&thr[i], NULL, async_zero, &thr_data[i]))) {
                fprintf(stderr, "error: pthread_create, rc: %d\n", result);
            }
        }
        uint64_t free = 0, total = 0;
        CUDA_SAFECALL(hipMemGetInfo(&free, &total));
        host_metadata_len = roundUp(total, GRAN);
        /* UVM ensures lazy allocation at 64K boundaries. Below allocations create a hash map for all posisble locations present on
           the GPU. Being lazily allocated, it does not consume the whole GPU memory area even though the VA space is quite large. */
        hipMallocManaged((void**)&device_arguments.memory_meta, sizeof(uint32_t) * host_metadata_len);
        if (DO_STREAM)
            hipMallocManaged((void**)&device_arguments.stream_meta, sizeof(uint32_t) * host_metadata_len * NUM_STREAM_TRACES);
        device_arguments.length = host_metadata_len;
        access_map = new std::atomic<uint64_t>[host_metadata_len];
        /* creating high priority stream for prefetching, async memset and memcpy */
        int high, low;
        hipDeviceGetStreamPriorityRange(&low, &high);
        hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, high);
        /* Initialize global variables as well  */
        static_counter = 0;
        allocation_records.clear();
        last_job.exchange(0);
    }
    setup.end();
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (!recv_thread_started)
        return;

    recv_thread_started = false;
    pthread_join (recv_thread, NULL);
    pthread_join (async_task, NULL);
    /* Wait till all worker threads are done */
    for (int i = 0; i < NUM_THREADS; i++) {
        //if (verbose) {
        //    printf("Joined: %d, with status: %d\n", i, pthread_join(thr[i], NULL));
        //}
        pthread_join(thr[i], NULL);
    }

    if (DO_ANALYZE) {
        /* Print suggestions */
        printf("========== SUGGESTIONS ==========\n");
        for (int i = 0; i < epoch; i++) {
            auto current = fence_map[i];
            if (!current->not_oversynchronized.load()) {
                uint64_t addr = id_to_fence_map[i];
                auto next = fence_map[i+1];
                /* NOTE: wrapper script depends on this format. Do not change without changing the wrapper! */
                printf("Fence@: %lx | Epoch: %d | Info: %s | Type: %s\n", addr, i, fence_to_lineinfo_map[addr].c_str(),
                    fence_map[i]->get_comment(next->operations.load()).c_str());
            }
        }
    }
    printCounters();
    printTrackers();
}
