#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdint.h>
#include <stdio.h>

// Comment to remove printfs
//#define DEBUG

#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"

__device__ __inline__
void dev_sleep(int &delay) {
    if(delay) {
        csleep(delay);
        /* Architecture dependent instruction. Available from volta onwards. */
        //__nanosleep(delay);
        delay *= 2;
        delay = min(delay, MAX_DELAY);
    }
}


__device__ __inline__
void set_device_metadata(uint64_t &metadata, uint32_t op_mask, uint64_t bid) {
    uint64_t old_id = getBits(metadata, POS_ID, SZ_ID);
    uint64_t first = getBit(metadata, POS_F);
    /* This is the important information */
    if (bid != old_id && first != 0) {
        setBit(metadata, POS_MB);
    }
    setBit(metadata, POS_F);

    uint64_t old_st = getBit(metadata, POS_ST);
    setBit(metadata, POS_ST, old_st | (op_mask & MASK_STORE));

    setBits(metadata, POS_ID, SZ_ID, bid);
    // This metadata also houses 'count' of traces present in stream_meta.
    // The field is updated in send_trace method.
}


__device__ __inline__
uint64_t set_host_metadata(long id, int epoch, uint32_t op_mask) {
    uint64_t info = 0;
    setBit(info, HPOS_LD, MASK_LOAD & op_mask);
    setBit(info, HPOS_ST, MASK_STORE & op_mask);
    setBits(info, HPOS_SCP, HSZ_SCP, (op_mask & SCOPE_CTA) | (op_mask & SCOPE_GPU) | (op_mask & SCOPE_SYS));
    setBits(info, HPOS_ID, HSZ_ID, id);
    setBits(info, HPOS_EP, HSZ_EP, epoch);
    return info;
}


__device__ __inline__
bool skip_trace(uint32_t op_mask) {
    bool skip = false;
    if (DO_FILTER) {
        // First two bits in mask is the scope of the operation, get it!
        int scp = (op_mask & 3);
        /* A Load and with scope greater than equal to device is enough
           for a volatile load and atomics with device_scope or larger */
        if ((MASK_LOAD & op_mask) && (scp >= SCOPE_GPU))
            skip = true;
    }
    return skip;
}

__device__ __inline__
bool skip_instrumentation(dev_args *dev, uint64_t global_tid, uint64_t global_bid, int instr) {
    bool skip = true;
    long dimension = dev->threads;
    /* Location of pointer, is of char pointer */;
    char *instr_meta = dev->sampling_meta;

    char local = instr_meta[dimension * instr + global_tid];
    /* Once every PER_THREAD_PER_INSTR instructions */
    if (local == 0  || local == dev->random_meta[global_bid])
        skip = false;
    local += 1;
    if (local > PER_THREAD_PER_INSTR)
        local = SAMP_BASE;
    instr_meta[dimension * instr + global_tid] = local;
    return skip;
}

/*
 * This method performs a number of operations and runs optimizations
 * 1. Update the in-GPU metadata maintained per address (md_up)
 * 2. Filters trace based on the type of operation (skip_trace)
 * 3. Maintain some content on the GPU {a.k.a. streaming access-type content}
 */
__device__ __inline__
bool send_trace(dev_args *dev, uint64_t offset, uint64_t tid, int epoch, uint32_t op_mask, uint64_t &md_up, uint64_t bid) {
    /* first set up content inside GPU aggregate metadata */
    set_device_metadata(md_up, op_mask, bid);
    /* return value */
    bool should_trace = false;
    // Do not maintain trace in stream_meta (version 1) if it does not help in detection
    if (!skip_trace(op_mask)) {
        uint64_t count = getBits(md_up, POS_CNT, SZ_CNT);
        if (DO_STREAM && count < NUM_STREAM_TRACES) {
            // prepare the trace that needs recording
            uint64_t trace = set_host_metadata(tid, epoch, op_mask);
            if (count != 0) {
                // some traces exist, write to a new position
                offset = offset + count * dev->length;
            }
            uint32_t *stream_meta = dev->stream_meta;
            stream_meta[offset] = trace;
            /* update count in aggregate metadata */
            count += 1;
            setBits(md_up, POS_CNT, SZ_CNT, count);
        } else {
            // no place in stream_meta, send it to host!
            should_trace = true;
        }
    }
    return should_trace;
}

extern "C" __device__ __noinline__
void instrument_fence(int pred, uint32_t fenceId, uint64_t args) {
#if DO_ANALYZE
    if (!pred)
        return;

    unsigned mask = __activemask();
    unsigned selectedThread = ((mask - 1) & mask) ^ mask;
    uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);

    /* Only 1 thread within active ones sends information */
    if (selectedThread & (1 << (tid % WARP_SIZE))) {
        dev_args *dev = (dev_args *)args;

        uint64_t wid = tid / WARP_SIZE;
        uint64_t warps_per_blk = roundUp(blockDim.x * blockDim.y * blockDim.z, WARP_SIZE);
        uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
        /* global warpid */
        wid = wid + bid * warps_per_blk;

        /* set fence metadata */
        uint64_t idx = dev->warps_per_grid * fenceId + wid;
        atomicOr(&dev->fence_meta[idx], mask);
    }

    __syncwarp(mask);
#endif
}

/* Tracing memory accesses by each thread
   1. predicate - guard predicate for the instruction
   2. addr - virtual address accessed by the instruciton
   3. op_mask - load/store/scope of operation
   4. epoch - for further analysis
 */
extern "C" __device__ __noinline__
void instrument_mem(int pred, uint64_t addr, uint32_t op_mask, volatile int epoch, uint32_t size, uint32_t instr, uint64_t args) {
#if DO_ANALYZE
    if (!pred)
        return;

    // Check if address belongs to global memory using PTX
    int is_global_mem;
    asm (".reg .pred p;\
        isspacep.global  p, %1;\
        selp.u32 %0,1,0,p;\
        ":"=r"(is_global_mem): "l"(addr));

    if(is_global_mem) {
        unsigned mask = __activemask();
        dev_args *dev = (dev_args *)args;

        // threadId -- global
        uint64_t tid = serializeId(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z);
        uint64_t bid = serializeId(blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
        tid = tid + bid * dev->threads_per_block;

        /* Skip: Execution sampling */
        if (DO_SAMPLING && skip_instrumentation(dev, tid, bid, instr)) {
            /* non-sampled instance, do nothing */
        } else {
            uint32_t *md_array = dev->memory_meta;
            uint64_t len = dev->length;
            int offset = 0, delay = BASE_DELAY;

            do {
                uint64_t md_offset = (addr + offset) / GRAN;
                md_offset = md_offset % len;
                unsigned int* md_addr = &(md_array)[md_offset];
                uint32_t md = atomicAdd(md_addr, 0);
                /* Need to lock before updating metadata, custom locking method */
                if (md == D_LOCKED) {
                    dev_sleep(delay);
                    continue;
                }

                if (atomicCAS(md_addr, md, D_LOCKED) == md) {
                    __threadfence();
                    uint64_t md_up = md;
                    /* should trace be tracked? */
                    bool trace = send_trace(dev, md_offset, tid, epoch, op_mask, md_up, bid);
                    md = md_up;
                    __threadfence();
                    /* update GPU metadata */
                    atomicExch(md_addr, md);
                    /* send the trace after releasing the lock */
                    if (trace) {
                        mem_access_t ma;
                        ma.addr = addr + offset;
                        ma.info = set_host_metadata(tid, epoch, op_mask);

                        channel_t c;
                        c.type = TYPE_MEM;
                        c.ma = ma;
                        ChannelDev *cdev = dev->channel_dev;
                        cdev->push (&c, sizeof(channel_t));
                    }
                    /* recorded meta, go to next size-offset */
                    offset += GRAN;
                    /* reset backoff delay for the next offset */
                    delay = BASE_DELAY;
                } else {
                    dev_sleep(delay);
                }
            } while(offset < size);
        }

        /* sync */
        __syncwarp(mask);
    }
#endif
}
